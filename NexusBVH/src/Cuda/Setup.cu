#include "hip/hip_runtime.h"
#include "Setup.h"

#include <cub/device/device_radix_sort.cuh>
#include <>

#include "CudaUtils.h"
#include "BuilderUtils.h"

__global__ void NXB::ComputePrimBounds(BuildState buildState, Triangle* primitives)
{
	uint32_t primIdx = blockDim.x * blockIdx.x + threadIdx.x;

	if (primIdx >= buildState.primCount)
		return;

	Triangle triangle = primitives[primIdx];

	buildState.primBounds[primIdx] = triangle.Bounds();
}

__global__ void NXB::ComputeSceneBounds(BuildState buildState)
{
	uint32_t primIdx = blockDim.x * blockIdx.x + threadIdx.x;

	// Make sure to initialize scene bounds
	if (primIdx == 0)
		buildState.sceneBounds->Clear();

	__syncthreads();

	if (primIdx >= buildState.primCount)
		return;

	// Shared bounds to parallelize atomic operations across thread blocks before updating the global scene bounds
	__shared__ AABB sharedBounds;

	// Clear shared AABB
	if (threadIdx.x == 0)
		sharedBounds.Clear();

	__syncthreads();

	AABB primBounds = buildState.primBounds[primIdx];
	AtomicGrow(&sharedBounds, primBounds);

	__syncthreads();

	// Scene bounds update
	if (threadIdx.x == 0)
		AtomicGrow(buildState.sceneBounds, sharedBounds);
}

__global__ void NXB::ComputeMortonCodes(BuildState buildState)
{
	uint32_t primIdx = blockDim.x * blockIdx.x + threadIdx.x;

	if (primIdx >= buildState.primCount)
		return;

	AABB primBounds = buildState.primBounds[primIdx];
	AABB *sceneBounds = buildState.sceneBounds;
	float3 centroid = primBounds.Centroid();

	// TODO: change division into mutliplication (store the inverse scene bounds)
	uint64_t mortonCode = MortonCode((centroid - sceneBounds->bMin) / (sceneBounds->bMax - sceneBounds->bMin));
	buildState.mortonCodes[primIdx] = mortonCode;

	// Initialize primitive indices as well
	buildState.primIdx[primIdx] = primIdx;
}

float NXB::RadixSort(BuildState& buildState)
{
	using byte = unsigned char;

	size_t tempStorageBytes = 0;
	void* tempStorage = nullptr;

	uint64_t* mortonCodesSorted = CudaMemory::AllocAsync<uint64_t>(buildState.primCount);
	uint32_t* primIdxSorted = CudaMemory::AllocAsync<uint32_t>(buildState.primCount);

	// Get the temporary storage size necessary to perform radix sorting
	hipcub::DeviceRadixSort::SortPairs(
		tempStorage,	// NULL
		tempStorageBytes,
		buildState.mortonCodes,
		mortonCodesSorted,
		buildState.primIdx,
		primIdxSorted,
		buildState.primCount,
		0,
		64
	);

	tempStorage = CudaMemory::AllocAsync<byte>(tempStorageBytes);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// Perform radix sorting
	hipcub::DeviceRadixSort::SortPairs(
		tempStorage,
		tempStorageBytes,
		buildState.mortonCodes,
		mortonCodesSorted,
		buildState.primIdx,
		primIdxSorted,
		buildState.primCount,
		0,
		64
	);

	float elapsedTime;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	CudaMemory::FreeAsync(tempStorage);

	CudaMemory::FreeAsync(buildState.mortonCodes);
	CudaMemory::FreeAsync(buildState.primIdx);

	buildState.mortonCodes = mortonCodesSorted;
	buildState.primIdx = primIdxSorted;

	return elapsedTime;
}

__global__ void NXB::InitClusters(BuildState buildState)
{
	const uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < buildState.primCount)
	{
		// Initialize first N leaf nodes
		BVH2::Node node;
		node.bounds = buildState.primBounds[buildState.primIdx[idx]];
		node.leftChild = INVALID_IDX;
		node.rightChild = buildState.primIdx[idx];
		buildState.nodes[idx] = node;

		// Initialize cluster indices to leaf node indices
		buildState.clusterIdx[idx] = idx;
	}
}
